
#include <hip/hip_runtime.h>
#include <stdio.h>

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

__global__ void mykernel(){
  printf("Hello from block %d, thread %d.\n", blockIdx.x, threadIdx.x);
}

int main(){

  mykernel<<<2,2>>>();
  hipDeviceSynchronize();
  cudaCheckErrors("kernel fail");
  return 0;
}
